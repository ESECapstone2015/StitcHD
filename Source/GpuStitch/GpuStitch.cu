#include "hip/hip_runtime.h"
/*
This file is part of StitcHD.

StitcHD is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

StitcHD is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with StitcHD.  If not, see <http://www.gnu.org/licenses/>.
*/

#define CHANNELS 3
#define MAX_CAMERAS 4

#include "GpuStitch.h"


#include "math_functions.h"
#include "hip/hip_runtime.h"
#include "common_functions.h"

#include <limits>
#include <ctime>
using namespace std;

#include <iostream>
#include <opencv2/gpu/devmem2d.hpp>
#include <opencv2/gpu/gpumat.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace cv::gpu;

namespace GpuStitch
{
	typedef unsigned char Tpixel;
	typedef float Thmg;

	const int MaxFrames = 4;

	// Used in device threads to transform a point using a homography
	__device__
	void applyHomographyToPoint(const int& x0, const int& y0,
		const DevMem2D_<Thmg>& hmgMem,
		float& x1, float& y1)
	{
		Thmg x, y, z;
		z = hmgMem.ptr(2)[0] * x0;
		z += hmgMem.ptr(2)[1] * y0;
		z += hmgMem.ptr(2)[2];

		x = hmgMem.ptr(0)[0] * x0;
		x += hmgMem.ptr(0)[1] * y0;
		x += hmgMem.ptr(0)[2];

		y = hmgMem.ptr(1)[0] * x0;
		y += hmgMem.ptr(1)[1] * y0;
		y += hmgMem.ptr(1)[2];

		x1 = x / z;
		y1 = y / z;
	}

	// Find the distance between two points
	__device__
	float getDistance(const int& x1, const int& y1,
		const int& x2, const int& y2)
	{
		return sqrtf( (x2 - x1)*(x2 - x1) + (y2 - y1)*(y2 - y1) );
	}

	// Used in interpolation to get the value of a Mat at a certain point
	__device__
	void getMatColorValue(const DevMem2D_<Tpixel>& src,
		const int& x, const int& y,
		int& val1, int& val2, int& val3)
	{
		if (x < 0 || x >= src.cols || y < 0 || y >= src.rows)
		{
			val1 = 0;
			val2 = 0;
			val3 = 0;
		}
		else
		{
			val1 = src.ptr(y)[x * CHANNELS];
			val2 = src.ptr(y)[x * CHANNELS + 1];
			val3 = src.ptr(y)[x * CHANNELS + 2];
		}
	}

	// Adds the value of a frame at a certain point on the canvas to the current RGB values at that point
	// Returns the multiplier for that point - depends on alpha blending
	__device__
	float addFrameToPixel(int& val1, int& val2, int& val3,
		const int& x, const int& y,
		const DevMem2D_<Tpixel>& src, const DevMem2D_<Thmg>& hmg,
		const StitchParams& params)
	{
		// Transform the pixel indices using the homography
		float tX, tY;
		applyHomographyToPoint(x, y, hmg, tX, tY);
		
		// Round them down to the nearest int
		int tXi = round(tX);
		int tYi = round(tY);

		if (!params.interpolate)
		{
			if (tXi < 1 || tXi >= src.cols || tYi < 1 || tYi >= src.rows)
				return 0;
		}
		else
		{
			if (tXi < 1 || tXi + 1 >= src.cols || tYi < 1 || tYi + 1 >= src.rows)
				return 0;
		}
		
		float rc;
		if (params.alphaBlend == 2)
		{
			// Linear blending
			rc = 1.0 - getDistance(tX, tY, src.cols / 2, src.rows / 2) / getDistance(0, 0, src.cols / 2, src.rows / 2);
		}
		else if (params.alphaBlend == 3)
		{
			// Exponential decay blending

			//		Get the Distance
			rc = getDistance(tX, tY, src.cols / 2, src.rows / 2) / getDistance(0, 0, src.cols / 2, src.rows / 2);
			//		Find the exponent
			rc = -(34.0 * params.expBlendValue + 100.0) * (rc - 0.5) / 50.0 - 1.0;
			//		Calculate the exponential
			rc = powf(10, rc);
		}
		else
		{
			// Average blending
			rc = 1.0;
		}

		if (!params.interpolate)
		{
			val1 += src.ptr(tYi)[tXi * CHANNELS];
			val2 += src.ptr(tYi)[tXi * CHANNELS + 1];
			val3 += src.ptr(tYi)[tXi * CHANNELS + 2];
			return rc;
		}
		else
		{
			// Bilinear interpolation

			tXi = floor(tX);
			tYi = floor(tY);

			float dX = tX - tXi;	// differential
			float dY = tY - tYi;

			int rgbValues[4][3];

			getMatColorValue(src, tXi, tYi,		rgbValues[0][0], rgbValues[0][1], rgbValues[0][2]);
			getMatColorValue(src, tXi+1, tYi,	rgbValues[1][0], rgbValues[1][1], rgbValues[1][2]);
			getMatColorValue(src, tXi, tYi+1,	rgbValues[2][0], rgbValues[2][1], rgbValues[2][2]);
			getMatColorValue(src, tXi+1, tYi+1,	rgbValues[3][0], rgbValues[3][1], rgbValues[3][2]);

			float x;

			// Update val1
			x = rgbValues[0][0] * (1-dX) * (1-dY) +
				rgbValues[1][0] * (dX) * (1-dY) +
				rgbValues[2][0] * (1-dX) * (dY) +
				rgbValues[3][0] * (dX) * (dY);
			if (x < 0)
				val1 = 0;
			else if (x > (unsigned char)(-1))
				val1 = (unsigned char)(-1);
			else
				val1 = x;

			// Update val2
			x = rgbValues[0][1] * (1-dX) * (1-dY) +
				rgbValues[1][1] * (dX) * (1-dY) +
				rgbValues[2][1] * (1-dX) * (dY) +
				rgbValues[3][1] * (dX) * (dY);
			if (x < 0)
				val2 = 0;
			else if (x > (unsigned char)(-1))
				val2 = (unsigned char)(-1);
			else
				val2 = x;

			// Update val3
			x = rgbValues[0][2] * (1-dX) * (1-dY) +
				rgbValues[1][2] * (dX) * (1-dY) +
				rgbValues[2][2] * (1-dX) * (dY) +
				rgbValues[3][2] * (dX) * (dY);
			if (x < 0)
				val3 = 0;
			else if (x > (unsigned char)(-1))
				val3 = (unsigned char)(-1);
			else
				val3 = x;

			return rc;
		}
	}

	// This is the entry point for the GPU - the kernel
	__global__
	void stitch_kernel(const int numFrames,
		DevMem2D_<Tpixel> const * const matSrc,
		DevMem2D_<Thmg> const * const matHmg,
		DevMem2D_<Tpixel> matDst,
		const StitchParams params)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
	
		if ((x < matDst.cols) && (y < matDst.rows))
		{
			int val1 = 0;	// RGB values
			int val2 = 0;
			int val3 = 0;

			unsigned char ucMax = (unsigned char)(-1);	// Comes in handy several times

			float multiplier = 0.0;
			float midV1=0, midV2=0, midV3=0;

			for (int i=0; i<numFrames; i++)
			{
				int v1=0, v2=0, v3=0;
				float m = addFrameToPixel(v1, v2, v3, x, y, matSrc[i], matHmg[i], params);

				if (m > 0.0)
				{
					if (params.hardShift)
					{
						switch (i)
						{
						case 0:
							v1 = ucMax; v2 = ucMax; v3 = ucMax;
							break;
						case 1:
							v1 = 0; v2 = 0; v3 = ucMax;
							break;
						case 2:
							v1 = 0; v2 = ucMax; v3 = 0;
							break;
						case 3:
							v1 = ucMax; v2 = 0; v3 = 0;
							break;
						default:
							break;	// We'll never get here
						}
					}
					else
					{
						if (i == 1) v3 += params.shift;
						if (i == 2) v2 += params.shift;
						if (i == 3) v1 += params.shift;
					}

					midV1 += m * v1;
					midV2 += m * v2;
					midV3 += m * v3;

					multiplier += m;
					//multiplier += 1.0;

					if (params.alphaBlend == 0)
						break;
				}
			}

			if (multiplier > 0.0)
			{
				val1 = int(midV1 / multiplier);
				val2 = int(midV2 / multiplier);
				val3 = int(midV3 / multiplier);

				if (val1 > ucMax)
					val1 = ucMax;
				if (val2 > ucMax)
					val2 = ucMax;
				if (val3 > ucMax)
					val3 = ucMax;
			}

			int pixelIdx = y * matDst.step + x * CHANNELS;
			matDst[pixelIdx] = val1;
			matDst[pixelIdx + 1] = val2;
			matDst[pixelIdx + 2] = val3;
		}
	}

	// Helpful function to check to see if a Cuda error has occurred
	__host__
	bool checkForCudaError(char* message)
	{
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			cout << "ERROR (" << message << "): " << hipGetErrorString(error) << endl;
			return true;
		}
		return false;
	}

	// Used in host thread to transform a point using a homography
	__host__
	Point applyHomographyToPoint(int x, int y, Mat &homography)
	{
		Point point;
		double z = (homography.at<double>(2, 0) * x + homography.at<double>(2, 1) * y + homography.at<double>(2, 2));
		double scale = 1./z;
		point.x = cvRound((homography.at<double>(0, 0) * x + homography.at<double>(0, 1) * y + homography.at<double>(0, 2)) * scale);
		point.y = cvRound((homography.at<double>(1, 0) * x + homography.at<double>(1, 1) * y + homography.at<double>(1, 2)) * scale);
		return point;
	}

	__host__
	Mat stitch_gpu(
		vector<Mat> matSrc,
		vector<Mat> matHmg,
		StitchParams params)
	{
		int numFrames = matSrc.size();

		if (numFrames < 2)
		{
			cout << "ERROR: Too few frames to stitch." << endl;
			return Mat(0,0,0);
		}
		if (numFrames > MaxFrames)
		{
			cout << "ERROR: Too many frames to stitch." << endl;
			return Mat(0,0,0);
		}
		if (numFrames != matHmg.size())
		{
			cout << "ERROR: Frames and Homographies don't match." << endl;
			return Mat(0,0,0);
		}
		for (int i=0; i<numFrames; i++)
		{
			if (matHmg[i].rows == 0 || matHmg[i].cols == 0)
				return Mat(0,0,0);
		}

		int minX = 0;
		int minY = 0; 
		int maxX = matSrc[0].cols;
		int maxY = matSrc[0].rows;

		for (int i=1; i<matSrc.size(); i++)
		{
			Mat inv = matHmg[i].inv();
			Point p = applyHomographyToPoint(0, 0, inv);
			if (p.x > maxX)		maxX = p.x;
			if (p.x < minX)		minX = p.x;
			if (p.y > maxY)		maxY = p.y;
			if (p.y < minY)		minY = p.y;
			p = applyHomographyToPoint(0, matSrc[i].rows - 1, inv);
			if (p.x > maxX)		maxX = p.x;
			if (p.x < minX)		minX = p.x;
			if (p.y > maxY)		maxY = p.y;
			if (p.y < minY)		minY = p.y;
			p = applyHomographyToPoint(matSrc[i].cols - 1, 0, inv);
			if (p.x > maxX)		maxX = p.x;
			if (p.x < minX)		minX = p.x;
			if (p.y > maxY)		maxY = p.y;
			if (p.y < minY)		minY = p.y;
			p = applyHomographyToPoint(matSrc[i].cols - 1, matSrc[i].rows - 1, inv);
			if (p.x > maxX)		maxX = p.x;
			if (p.x < minX)		minX = p.x;
			if (p.y > maxY)		maxY = p.y;
			if (p.y < minY)		minY = p.y;
		}

		int offsetX = matSrc[0].cols - (maxX + minX) / 2;
		int offsetY = matSrc[0].rows - (maxY + minY) / 2;

		Mat translation = (Mat_<double>(3,3) << 1, 0, -offsetX, 0, 1, -offsetY, 0, 0, 1);

		for(int i=0; i<matSrc.size(); i++)
			matHmg[i] = matHmg[i] * translation;

		GpuMat matDstDev = GpuMat(2 * matSrc[0].rows, 2 * matSrc[0].cols, CV_8UC3);

		vector<GpuMat> matSrcDev(MaxFrames);
		vector<GpuMat> matHmgDev(MaxFrames);

		DevMem2D_<Tpixel> matSrcMem_h[MAX_CAMERAS];
		DevMem2D_<Thmg> matHmgMem_h[MAX_CAMERAS];

		// Convert Mats into GpuMats and then DevMem2Ds
		for (int i=0; i<numFrames; i++)
		{
			matSrcDev[i] = GpuMat(matSrc[i]);
			Mat tmp;
			matHmg[i].convertTo(tmp, CV_32FC1);
			matHmgDev[i] = GpuMat(tmp);

			matSrcMem_h[i] = matSrcDev[i];
			matHmgMem_h[i] = matHmgDev[i];
		}

		void* matSrcMem_d;
		hipMalloc(&matSrcMem_d, sizeof(matSrcMem_h));

		if (checkForCudaError("hipMalloc"))
			return Mat(0,0,0);

		void* matHmgMem_d;
		hipMalloc(&matHmgMem_d, sizeof(matHmgMem_h));

		if (checkForCudaError("hipMalloc"))
		{
			hipFree(matHmgMem_d);
			return Mat(0,0,0);
		}

		hipMemcpy(matSrcMem_d, matSrcMem_h, sizeof(matSrcMem_h), hipMemcpyHostToDevice);

		if (checkForCudaError("hipMemcpy"))
		{
			hipFree(matSrcMem_d);
			hipFree(matHmgMem_d);
			return Mat(0,0,0);
		}

		hipMemcpy(matHmgMem_d, matHmgMem_h, sizeof(matHmgMem_h), hipMemcpyHostToDevice);

		if (checkForCudaError("hipMemcpy"))
		{
			hipFree(matSrcMem_d);
			hipFree(matHmgMem_d);
			return Mat(0,0,0);
		}

		dim3 block(32, 16, 1);

		int x = int(0.5f + float(matDstDev.cols * CHANNELS) / float(block.x));
		int y = int(0.5f + float(matDstDev.rows) / float(block.y));
		dim3 grid(x, y, 1);

		stitch_kernel<<<grid, block>>>(
			numFrames,
			(DevMem2D_<Tpixel>*)matSrcMem_d,
			(DevMem2D_<Thmg>*)matHmgMem_d,
			matDstDev,
			params);

		hipDeviceSynchronize();
		bool rc = checkForCudaError("after kernel launch");
		
		hipFree(matSrcMem_d);
		hipFree(matHmgMem_d);

		if (rc)
			return Mat(0,0,0);
		else
		{
			Mat output;
			matDstDev.download(output);
			return output;
		}
	}
}
